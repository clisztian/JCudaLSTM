/*
 * JCuda - Java bindings for NVIDIA CUDA driver and runtime API
 * http://www.jcuda.org
 *
 *
 * This code is based on the NVIDIA 'reduction' CUDA sample,
 * Copyright 1993-2010 NVIDIA Corporation.
 */


#include <hip/hip_runtime.h>
#include <math.h>
extern "C"
__global__ void reduce(double *g_idata, double *g_odata, unsigned int n)
{
    extern __shared__ double sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
    unsigned int gridSize = blockDim.x*2*gridDim.x;

    double mySum = 0;
  

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];
        // ensure we don't read out of bounds
        if (i + blockDim.x < n)
            mySum += g_idata[i+blockDim.x];
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockDim.x >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
        if (blockDim.x >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
        if (blockDim.x >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
        if (blockDim.x >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
        if (blockDim.x >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
        if (blockDim.x >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}